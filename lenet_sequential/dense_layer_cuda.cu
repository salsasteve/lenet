#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

template <const int BM, const int BN, const int BK, const int TM>
__global__ void sgemm_blocktiling_1d_kernel(float *A, float *B, float *C, int M, int N, int K)
{
    // the output block that we want to compute in this threadblock
    const int c_row = blockIdx.y;
    const int c_col = blockIdx.x;

    // allocate shared memory for the input and output submatrices
    __shared__ float A_shared[BM * BK];
    __shared__ float B_shared[BK * BN];

    // the inner row & col that we're accessing in this thread
    const int thread_row = threadIdx.x / BN;
    const int thread_col = threadIdx.x % BN;

    // advance pointers to the starting positions
    A += c_row * BM * K;
    B += c_col * BN;
    C += c_row * BM * N + c_col * BN;

    // use to avoid out-of-bounds accesses
    int global_m_pos = c_row * BM * K;
    int global_n_pos = c_col * BN;
    const int m_size = M * K;
    const int n_size = N * K;

    const int A_inner_row = threadIdx.x / BK; // warp-level GMEM coalescing
    const int A_inner_col = threadIdx.x % BK;
    const int B_inner_row = threadIdx.x / BN; // warp-level GMEM coalescing
    const int B_inner_col = threadIdx.x % BN;

    // allocate thread-local cache for results in registerfile
    float thread_results[TM] = {0.0};

    // outer loop over block tiles
    for (int bk_idx = 0; bk_idx < K; bk_idx += BK)
    {
        // load the next block of the input matrices into shared memory
        A_shared[A_inner_row * BK + A_inner_col] = (global_m_pos + A_inner_row * K + A_inner_col < m_size) ? A[A_inner_row * K + A_inner_col] : 0.0f;
        B_shared[B_inner_row * BN + B_inner_col] = (global_n_pos + B_inner_row * N + B_inner_col < n_size) ? B[B_inner_row * N + B_inner_col] : 0.0f;

        // wait for all threads to finish loading
        __syncthreads();

        // advance the pointers
        A += BK;
        B += BK * N;
        global_m_pos += BK;
        global_n_pos += BK * N;

        // compute the partial sum
        for (int dot_idx = 0; dot_idx < BK; dot_idx++)
        {
            // we make the dotproduct loop the outside loop, which facilitates
            // reuse of the Bs entry, which we can cache in a tmp var.
            float tmp_b = B_shared[dot_idx * BN + thread_col];
            for (int res_idx = 0; res_idx < TM; res_idx++)
            {
                thread_results[res_idx] += A_shared[(thread_row * TM + res_idx) * BK + dot_idx] * tmp_b;
            }
        }

        // wait for all threads to finish computing
        __syncthreads();
    }

    for (int res_idx = 0; res_idx < TM; res_idx++)
    {
        if (c_row * BM + thread_row * TM + res_idx < M && c_col * BN + thread_col < N)
        {
            C[(thread_row * TM + res_idx) * N + thread_col] = thread_results[res_idx];
        }
    }
}

__global__ void add_kernel(float *x, float *y, int n, bool activate){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < n) {
        if (activate){
            x[tid] = tanhf(x[tid] + y[tid]);
        }else{
            x[tid] = x[tid] + y[tid];
        }
    }
}


void run_sgemm_blocktiling_1d(float *A, float *B, float *bias, float *C, int m, int n, int k, bool activate)
{
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    dim3 grid_size(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
    dim3 block_size((BM * BN) / TM);
    sgemm_blocktiling_1d_kernel<BM, BN, BK, TM>
        <<<grid_size, block_size>>>(A, B, C, m, n, k);
    const int block = 256;
    const int grid = (n + block) / block;
    add_kernel<<<grid, block>>>(C, bias, n, activate);
}


std::vector<float> dense_GPU(
    std::vector<float> &input,
    std::vector<float> &biases,
    std::vector<std::vector<float>> &weights,
    int numOutputs,
    bool activate)
{
    int m = 1;
    int n = numOutputs;
    int k = weights.size();

    // Allocate memory for matrices
    float *A, *B, *C, *bias;
    float *d_A, *d_B, *d_C, *d_bias;

    A = new float[m * k];
    B = new float[k * n];
    C = new float[m * n];
    bias = new float[m * n];
    int index = 0;
    for (int i = 0; i < k; ++i){
        A[index++] = input[i];
    }
    index = 0;
    for (int i = 0; i < k; ++i){
        for (int j=0; j<n;j++){
            B[index++] = weights[i][j];
        }
    }
    index = 0;
    for (int i = 0; i < n; ++i){
        bias[index++]=biases[i];
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));
    hipMalloc((void **)&d_bias, m * n * sizeof(float));

    // Copy matrices to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, m * n * sizeof(float), hipMemcpyHostToDevice);

    run_sgemm_blocktiling_1d(d_A, d_B, d_bias, d_C, m, n, k, activate);

    // Copy result to host
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> output(n);
    index = 0;
    for (int i = 0; i < n; ++i){
        outputMaps[i] = out[index++];
    }
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_bias);
    free(A);
    free(B);
    free(C);
    free(bias);
    return output;
}
