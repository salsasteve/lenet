#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
using namespace std;
/*
 * @param n: batch size
 * @param c: number of channel
 * @param h: height
 * @param w: width
 * @param k: number of kernel
 * @param r: kernel height
 * @param s: kernel width
 * @param out_h: output height
 * @param out_w: output width
 * @param u: stride vertical
 * @param v: stride horizontal
 * @param p: padding height
 * @param q: padding width
 * @param in: input
 * @param weight: kernel
 * @param out: output
 */
__global__ void
naive_conv2d_kernel(int n, int c, int h, int w,
                    int k, int r, int s,
                    int out_h, int out_w,
                    int u, int v, int p, int q,
                    float *in, float *weight, float *bias, float *out)
{
    // 获取线程在三维网格中的位置
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z;

    // 如果线程超出工作范围则退出
    if (x >= out_h * out_w || y >= k || z >= n)
    {
        return;
    }

    // 当前线程处理的数据点在out_h、out_w上的坐标
    int pos_out_h = x / out_w;
    int pos_out_w = x % out_w;

    // 计算输入数据的坐标
    int pos_ori_h = pos_out_h * u - p;
    int pos_ori_w = pos_out_w * v - q;

    float sum = 0.0;

    int in_offset = z * c * h * w + pos_ori_h * w + pos_ori_w;
    int weight_offset = y * c * r * s;
    int in_channel_offset = h * w;
    int weight_channel_offset = r * s;

    // 执行卷积操作
    for (int i = 0; i < r; i++)
    {
        for (int j = 0; j < s; j++)
        {
            int pos_real_h = pos_ori_h + i;
            int pos_real_w = pos_ori_w + j;

            // 只处理有效的数据点
            if (pos_real_h >= 0 && pos_real_w >= 0 && pos_real_w < w && pos_real_h < h)
            {
                int in_offset_tmp = in_offset;
                int wei_offset_tmp = weight_offset;
                for (int channel = 0; channel < c; channel++)
                {
                    // 计算卷积和
                    sum += in[in_offset_tmp + i * w + j] * weight[wei_offset_tmp + i * s + j];
                    in_offset_tmp += in_channel_offset;
                    wei_offset_tmp += weight_channel_offset;
                }
            }
        }
    }

    // 计算输出偏移
    int out_offset = z * k * out_h * out_w + y * out_h * out_w + x;
    out[out_offset] = tanhf(sum+bias[y]);
}

void conv2d_gpu(const int n,
                const int c,
                const int h,
                const int w,
                const int k,
                const int r,
                const int s,
                const int u,
                const int v,
                const int p,
                const int q,
                const int out_h, // 输出高
                const int out_w, // 输出宽
                float *in,
                float *weight,
                float *bias,
                float *out
){
    float *in_device, *weight_device, *bias_device, *out_device;

    hipMalloc((void **)&in_device, n * c * h * w * sizeof(float));
    hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void **)&bias_device, k * sizeof(float));
    hipMalloc((void **)&out_device, n * k * out_h * out_w * sizeof(float));

    // 将输入数据和卷积核拷贝到 GPU
    hipMemcpy(in_device, in, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_device, bias, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_device, out, n * k * out_h * out_w * sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块的大小
    const int blockDim_x = 16;
    const int blockDim_y = 16;

    // 计算线程块和网格的数量
    const int gridDim_x = (out_h * out_w + blockDim_x - 1) / blockDim_x;
    const int gridDim_y = (k + blockDim_y - 1) / blockDim_y;

    // 定义线程块和网

    dim3 blockDim(blockDim_x, blockDim_y);
    dim3 gridDim(gridDim_x, gridDim_y, n);

    // 调用 kernel 函数
    naive_conv2d_kernel<<<gridDim, blockDim>>>(n, c, h, w, k, r, s, out_h, out_w, u, v, p, q, in_device, weight_device, bias_device, out_device);
    // 同步
    hipDeviceSynchronize();

    // 将 GPU 计算的结果拷贝到 CPU
    hipMemcpy(out, out_device, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(in_device);
    hipFree(weight_device);
    hipFree(bias_device);
    hipFree(out_device);
}

vector<vector<vector<float>>> convolve2dDeep_GPU(
    const vector<vector<vector<float>>> &inputMaps,
    const vector<vector<vector<vector<float>>>> &kernels,
    const vector<float> &biases,
    const int stride,
    const int padding){

    const int n = 1;                           // batch size
    const int c = inputMaps.size();                           // 通道数
    const int h = inputMaps[0].size();                          // 数据高
    const int w = inputMaps[0][0].size();                          // 数据宽
    const int f = kernels.size();                           // 卷积核数量
    const int r = kernels[0][0].size();                           // 卷积核高
    const int s = kernels[0][0][0].size();                           // 卷积核宽
    const int u = stride;                           // 卷积在高方向上的步长
    const int v = stride;                           // 卷积在宽方向上的步长
    const int p = padding;                           // 卷积在高方向上的补边
    const int q = padding;                           // 卷积在宽方向上的补边
    const int out_h = (h + 2 * p - r) / u + 1; // 输出高
    const int out_w = (w + 2 * q - s) / v + 1; // 输出宽

    float *in, *weight, *out, *bias;
    in = (float *)malloc(n * c * h * w * sizeof(float));
    weight = (float *)malloc(f * c * r * s * sizeof(float));
    bias = (float *)malloc(f *sizeof(float));
    out = (float *)malloc(n * f * out_h * out_w * sizeof(float));

    int index = 0;
    for (int i = 0; i < c; ++i){
        for (int j=0; j<h;j++){
            for (int k=0; k<w;k++){
                in[index++] = inputMaps[i][j][k];
            }
        }
    }
    index = 0;
    for (int i = 0; i < f; ++i){
        for (int j=0; j<c;j++){
            for (int k=0; k<r;k++){
                for (int l=0; l<s;l++){
                    weight[index++] = kernels[i][j][k][l];
                }
            }
        }
    }
    index = 0;
    for (int i = 0; i < f; ++i){
        bias[index++]=biases[i];
    }
    conv2d_gpu(n,                           // batch size
                c,                          // 通道数
                h,                          // 数据高
                w,                          // 数据宽
                f,                           // 卷积核数量
                r,                           // 卷积核高
                s,                           // 卷积核宽
                u,                           // 卷积在高方向上的步长
                v,                           // 卷积在宽方向上的步长
                p,                           // 卷积在高方向上的补边
                q,                           // 卷积在宽方向上的补边
                out_h,                       // 输出高
                out_w,                       // 输出宽
                in, weight, bias, out);
    // Initialize output maps
    vector<vector<vector<float>>> outputMaps(f, vector<vector<float>>(out_h, vector<float>(out_w, 0.0)));

    // Perform convolution on each output feature map
    std::cout << "Biases count: " << biases.size() << endl;
    std::cout << "Kernels count: " << kernels.size() << endl;
    index = 0;
    for (int i = 0; i < f; ++i){
        for (int j=0; j<out_h;j++){
            for (int k=0; k<out_w;k++){
                outputMaps[i][j][k] = out[index++];
            }
        }
    }
    free(in);
    free(weight);
    free(bias);
    free(out);
    return outputMaps;
}