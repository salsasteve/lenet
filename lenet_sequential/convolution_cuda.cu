#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
using namespace std;
/*
 * @param n: batch size
 * @param c: number of channel
 * @param h: height
 * @param w: width
 * @param k: number of kernel
 * @param r: kernel height
 * @param s: kernel width
 * @param out_h: output height
 * @param out_w: output width
 * @param u: stride vertical
 * @param v: stride horizontal
 * @param p: padding height
 * @param q: padding width
 * @param in: input
 * @param weight: kernel
 * @param out: output
 */
__global__ void
naive_conv2d_kernel(int n, int c, int h, int w,
                    int k, int r, int s,
                    int out_h, int out_w,
                    int u, int v, int p, int q,
                    float *in, float *weight, float *bias, float *out)
{
    // 获取线程在三维网格中的位置
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z;

    // 如果线程超出工作范围则退出
    if (x >= out_h * out_w || y >= k || z >= n)
    {
        return;
    }

    // 当前线程处理的数据点在out_h、out_w上的坐标
    int pos_out_h = x / out_w;
    int pos_out_w = x % out_w;

    // 计算输入数据的坐标
    int pos_ori_h = pos_out_h * u - p;
    int pos_ori_w = pos_out_w * v - q;

    float sum = 0.0;

    int in_offset = z * c * h * w + pos_ori_h * w + pos_ori_w;
    int weight_offset = y * c * r * s;
    int in_channel_offset = h * w;
    int weight_channel_offset = r * s;

    // 执行卷积操作
    for (int i = 0; i < r; i++)
    {
        for (int j = 0; j < s; j++)
        {
            int pos_real_h = pos_ori_h + i;
            int pos_real_w = pos_ori_w + j;

            // 只处理有效的数据点
            if (pos_real_h >= 0 && pos_real_w >= 0 && pos_real_w < w && pos_real_h < h)
            {
                int in_offset_tmp = in_offset;
                int wei_offset_tmp = weight_offset;
                for (int channel = 0; channel < c; channel++)
                {
                    // 计算卷积和
                    sum += in[in_offset_tmp + i * w + j] * weight[wei_offset_tmp + i * s + j];
                    in_offset_tmp += in_channel_offset;
                    wei_offset_tmp += weight_channel_offset;
                }
            }
        }
    }

    // 计算输出偏移
    int out_offset = z * k * out_h * out_w + y * out_h * out_w + x;
    out[out_offset] = tanhf(sum+bias[y]);
}

// CPU 端的卷积计算
void conv2d_cpu(float *in, float *pwei, float *bias, float *out, int n, int c, int h, int w, int k, int r, int s, int u, int v, int p, int q, int out_h, int out_w)
{

    for (int n_num = 0; n_num < n; n_num++)
    {
        for (int k_num = 0; k_num < k; k_num++)
        {
            for (int i = 0; i < out_h; i++)
            {
                for (int j = 0; j < out_w; j++)
                {
                    double sum = 0.0;
                    int pos_h = i * u - p;
                    int pos_w = j * v - q;

                    for (int c_num = 0; c_num < c; c_num++)
                    {
                        for (int kh_num = 0; kh_num < r; kh_num++)
                        {
                            for (int kwNum = 0; kwNum < s; kwNum++)
                            {
                                int pos_ori_h = pos_h + kh_num;
                                int pos_ori_w = pos_w + kwNum;
                                if (pos_ori_w >= 0 && pos_ori_h >= 0 && pos_ori_w < w && pos_ori_h < h)
                                {
                                    sum += (double)(in[n_num * c * h * w + c_num * (w * h) + pos_ori_h * w + pos_ori_w] * pwei[k_num * r * s * c + c_num * r * s + kh_num * s + kwNum]);
                                }
                            }
                        }
                    }

                    out[n_num * k * out_h * out_w + k_num * out_h * out_w + i * out_w + j] = tanhf((float)sum+bias[k_num]);
                }
            }
        }
    }
}

void conv2d_gpu(const int n,
                const int c,
                const int h,
                const int w,
                const int k,
                const int r,
                const int s,
                const int u,
                const int v,
                const int p,
                const int q,
                const int out_h, // 输出高
                const int out_w, // 输出宽
                float *in,
                float *weight,
                float *bias,
                float *out
){
    float *in_device, *weight_device, *bias_device, *out_device;

    hipMalloc((void **)&in_device, n * c * h * w * sizeof(float));
    hipMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void **)&bias_device, k * sizeof(float));
    hipMalloc((void **)&out_device, n * k * out_h * out_w * sizeof(float));

    // 将输入数据和卷积核拷贝到 GPU
    hipMemcpy(in_device, in, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_device, bias, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_device, out, n * k * out_h * out_w * sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块的大小
    const int blockDim_x = 16;
    const int blockDim_y = 16;

    // 计算线程块和网格的数量
    const int gridDim_x = (out_h * out_w + blockDim_x - 1) / blockDim_x;
    const int gridDim_y = (k + blockDim_y - 1) / blockDim_y;

    // 定义线程块和网

    dim3 blockDim(blockDim_x, blockDim_y);
    dim3 gridDim(gridDim_x, gridDim_y, n);

    // 调用 kernel 函数
    naive_conv2d_kernel<<<gridDim, blockDim>>>(n, c, h, w, k, r, s, out_h, out_w, u, v, p, q, in_device, weight_device, bias_device, out_device);
    // 同步
    hipDeviceSynchronize();

    // 将 GPU 计算的结果拷贝到 CPU
    hipMemcpy(out, out_device, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(in_device);
    hipFree(weight_device);
    hipFree(out_device);
}

vector<vector<vector<float>>> convolve2dDeep_GPU(
    const vector<vector<vector<float>>> &inputMaps,
    const vector<vector<vector<vector<float>>>> &kernels,
    const vector<float> &biases,
    const int stride,
    const int padding){

    const int n = 1;                           // batch size
    const int c = inputMaps.size();                           // 通道数
    const int h = inputMaps[0].size();                          // 数据高
    const int w = inputMaps[0][0].size();                          // 数据宽
    const int k = kernels.size();                           // 卷积核数量
    const int r = kernels[0][0].size();                           // 卷积核高
    const int s = kernels[0][0][0].size();                           // 卷积核宽
    const int u = stride;                           // 卷积在高方向上的步长
    const int v = stride;                           // 卷积在宽方向上的步长
    const int p = padding;                           // 卷积在高方向上的补边
    const int q = padding;                           // 卷积在宽方向上的补边
    const int out_h = (h + 2 * p - r) / u + 1; // 输出高
    const int out_w = (w + 2 * q - s) / v + 1; // 输出宽

    float *in, *weight, *out, *bias;
    in = (float *)malloc(n * c * h * w * sizeof(float));
    weight = (float *)malloc(k * c * r * s * sizeof(float));
    bias = (float *)malloc(k *sizeof(float));
    out = (float *)malloc(n * k * out_h * out_w * sizeof(float));

    int index = 0;
    for (int i = 0; i < c; ++i){
        for (int j=0; j<h;j++){
            for (int k=0; k<w;k++){
                in[index++] = inputMaps[i][j][k];
            }
        }
    }
    index = 0;
    for (int i = 0; i < k; ++i){
        for (int j=0; j<c;j++){
            for (int k=0; k<r;k++){
                for (int l=0; l<s;l++){
                    weight[index++] = kernels[i][j][k][l];
                }
            }
        }
    }
    index = 0;
    for (int i = 0; i < k; ++i){
        bias[index++]=biases[i];
    }
    conv2d_gpu(n,                           // batch size
                c,                          // 通道数
                h,                          // 数据高
                w,                          // 数据宽
                k,                           // 卷积核数量
                r,                           // 卷积核高
                s,                           // 卷积核宽
                u,                           // 卷积在高方向上的步长
                v,                           // 卷积在宽方向上的步长
                p,                           // 卷积在高方向上的补边
                q,                           // 卷积在宽方向上的补边
                out_h,                       // 输出高
                out_w,                       // 输出宽
                in, weight, bias, out);
    // Initialize output maps
    vector<vector<vector<float>>> outputMaps(k, vector<vector<float>>(out_h, vector<float>(out_w, 0.0)));

    // Perform convolution on each output feature map
    std::cout << "Biases count: " << biases.size() << endl;
    std::cout << "Kernels count: " << kernels.size() << endl;
    index = 0;
    for (int i = 0; i < k; ++i){
        for (int j=0; j<out_h;j++){
            for (int k=0; k<out_w;k++){
                outputMaps[i][j][k] = out[index++];
            }
        }
    }
    return outputMaps;
}

//
// int main()
// {
//     // 定义输入数据和卷积核的尺寸
//     const int n = 1;                           // batch size
//     const int c = 1;                           // 通道数
//     const int h = 28;                          // 数据高
//     const int w = 28;                          // 数据宽
//     const int k = 6;                           // 卷积核数量
//     const int r = 5;                           // 卷积核高
//     const int s = 5;                           // 卷积核宽
//     const int u = 1;                           // 卷积在高方向上的步长
//     const int v = 1;                           // 卷积在宽方向上的步长
//     const int p = 2;                           // 卷积在高方向上的补边
//     const int q = 2;                           // 卷积在宽方向上的补边
//     const int out_h = (h + 2 * p - r) / u + 1; // 输出高
//     const int out_w = (w + 2 * q - s) / v + 1; // 输出宽
//     // 分配内存并随机生成输入数据和卷积核
//     float *in, *weight, *bias, *out;
//     in = (float *)malloc(n * c * h * w * sizeof(float));
//     weight = (float *)malloc(k * c * r * s * sizeof(float));
//     bias = (float *)malloc(k *sizeof(float));
//     out = (float *)malloc(n * k * out_h * out_w * sizeof(float));
//     // 随机生成输入数据和卷积核
//     for (int i = 0; i < n * c * h * w; ++i)
//     {
//         in[i] = (float)rand() / RAND_MAX;
//     }
//     for (int i = 0; i < k * c * r * s; ++i)
//     {
//         weight[i] = (float)rand() / RAND_MAX;
//     }
//     for (int i = 0; i < k; ++i)
//     {
//         bias[i] = (float)rand() / RAND_MAX;
//     }
//     conv2d_gpu(n,                           // batch size
//                 c,                          // 通道数
//                 h,                          // 数据高
//                 w,                          // 数据宽
//                 k,                           // 卷积核数量
//                 r,                           // 卷积核高
//                 s,                           // 卷积核宽
//                 u,                           // 卷积在高方向上的步长
//                 v,                           // 卷积在宽方向上的步长
//                 p,                           // 卷积在高方向上的补边
//                 q,                           // 卷积在宽方向上的补边
//                 out_h,                       // 输出高
//                 out_w,                       // 输出宽
//                 in, weight, bias, out);
// //     float *in_device, *weight_device, *out_device;
// //
// //     cudaMalloc((void **)&in_device, n * c * h * w * sizeof(float));
// //     cudaMalloc((void **)&weight_device, k * c * r * s * sizeof(float));
// //     cudaMalloc((void **)&out_device, n * k * out_h * out_w * sizeof(float));
// //
// //
// //     // 将输入数据和卷积核拷贝到 GPU
// //     cudaMemcpy(in_device, in, n * c * h * w * sizeof(float), cudaMemcpyHostToDevice);
// //     cudaMemcpy(weight_device, weight, k * c * r * s * sizeof(float), cudaMemcpyHostToDevice);
// //     cudaMemcpy(out_device, out, n * k * out_h * out_w * sizeof(float), cudaMemcpyHostToDevice);
// //
// //     // 定义线程块的大小
// //     const int blockDim_x = 16;
// //     const int blockDim_y = 16;
// //
// //     // 计算线程块和网格的数量
// //     const int gridDim_x = (out_h * out_w + blockDim_x - 1) / blockDim_x;
// //     const int gridDim_y = (k + blockDim_y - 1) / blockDim_y;
// //
// //     // 定义线程块和网
// //
// //     dim3 blockDim(blockDim_x, blockDim_y);
// //     dim3 gridDim(gridDim_x, gridDim_y, n);
// //
// //     // 调用 kernel 函数
// //     naive_conv2d_kernel<<<gridDim, blockDim>>>(n, c, h, w, k, r, s, out_h, out_w, u, v, p, q, in_device, weight_device, out_device);
// //     // 同步
// //     cudaDeviceSynchronize();
// //
// //     // 将 GPU 计算的结果拷贝到 CPU
// //     cudaMemcpy(out, out_device, n * k * out_h * out_w * sizeof(float), cudaMemcpyDeviceToHost);
//
//     // CPU 端进行卷积计算
//     float *out_cpu = (float *)malloc(n * k * out_h * out_w * sizeof(float));
//     conv2d_cpu(in, weight, bias, out_cpu, n, c, h, w, k, r, s, u, v, p, q, out_h, out_w);
//
//     // 比较 GPU 和 CPU 计算结果是否一致
//     bool pass = true;
//     for (int i = 0; i < n * k * out_h * out_w; ++i)
//     {
//         if (abs(out[i] - out_cpu[i]) > 1e-5)
//         {
//             pass = false;
//             std::cout << "Verification failed at " << i << "!" << std::endl;
//             std::cout << "GPU: " << out_cpu[i] << " CPU: " << out[i] << std::endl;
//             break;
//         }
//     }
//     std::cout << "Verification Pass"<< std::endl;
// //     if (pass)
// //     {
// //         std::cout << "Verification passed!" << std::endl;
// //
// //         int iter = 100;
// //         cudaEvent_t start, stop;
// //         cudaEventCreate(&start);
// //         cudaEventCreate(&stop);
// //         cudaEventRecord(start, 0);
// //         for (int i = 0; i < iter; i++)
// //         {
// //             naive_conv2d_kernel<<<gridDim, blockDim>>>(n, c, h, w, k, r, s, out_h, out_w, u, v, p, q, in_device, weight_device, out_device);
// //         }
// //         cudaEventRecord(stop, 0);
// //         cudaEventSynchronize(stop);
// //         float elapsedTime;
// //         cudaEventElapsedTime(&elapsedTime, start, stop);
// //         std::cout << "GPU time: " << 1000 * elapsedTime / iter << "us" << std::endl;
// //         cudaEventDestroy(start);
// //         cudaEventDestroy(stop);
// //     }
//
//     // 释放内存
// //     cudaFree(in_device);
// //     cudaFree(weight_device);
// //     cudaFree(out_device);
//     free(in);
//     free(weight);
//     free(out);
//
//     return 0;
// }