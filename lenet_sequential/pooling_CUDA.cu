#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <math.h>
#include <vector>
using namespace std;
/*
 * @param n: batch size
 * @param c: number of channel
 * @param h: height
 * @param w: width
 * @param k: number of kernel
 * @param r: kernel height
 * @param s: kernel width
 * @param out_h: output height
 * @param out_w: output width
 * @param u: stride vertical
 * @param v: stride horizontal
 * @param p: padding height
 * @param q: padding width
 * @param in: input
 * @param weight: kernel
 * @param out: output
 */
__global__ void
naive_pooling_kernel(int n, int c, int h, int w,
                    int r, int s,
                    int out_h, int out_w,
                    int u, int v, int p, int q,
                    float *in, float *out)
{
    // 获取线程在三维网格中的位置
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int z = blockIdx.z;

    // 如果线程超出工作范围则退出
    if (x >= out_h * out_w || y >= c || z >= n)
    {
        return;
    }

    // 当前线程处理的数据点在out_h、out_w上的坐标
    int pos_out_h = x / out_w;
    int pos_out_w = x % out_w;

    // 计算输入数据的坐标
    int pos_ori_h = pos_out_h * u - p;
    int pos_ori_w = pos_out_w * v - q;

    float sum = 0.0;

    int in_offset = z * c * h * w + y * h * w + pos_ori_h * w + pos_ori_w;
    int in_channel_offset = h * w;

    // 执行卷积操作
    for (int i = 0; i < r; i++)
    {
        for (int j = 0; j < s; j++)
        {
            int pos_real_h = pos_ori_h + i;
            int pos_real_w = pos_ori_w + j;

            // 只处理有效的数据点
            if (pos_real_h >= 0 && pos_real_w >= 0 && pos_real_w < w && pos_real_h < h)
            {
                int in_offset_tmp = in_offset;
                // 计算卷积和
                sum += in[in_offset_tmp + i * w + j];
                in_offset_tmp += in_channel_offset;
            }
        }
    }

    // 计算输出偏移
    int out_offset = z * c * out_h * out_w + y * out_h * out_w + x;
    out[out_offset] = sum/ (r*s);
}

// CPU 端的卷积计算
void pooling_cpu(float *in, float *out, int n, int c, int h, int w, int r, int s, int u, int v, int p, int q, int out_h, int out_w)
{

    for (int n_num = 0; n_num < n; n_num++)
    {
        for (int c_num = 0; c_num < c; c_num++)
        {
            for (int i = 0; i < out_h; i++)
            {
                for (int j = 0; j < out_w; j++)
                {
                    double sum = 0.0;
                    int pos_h = i * u - p;
                    int pos_w = j * v - q;

                    for (int kh_num = 0; kh_num < r; kh_num++)
                    {
                        for (int kwNum = 0; kwNum < s; kwNum++)
                        {
                            int pos_ori_h = pos_h + kh_num;
                            int pos_ori_w = pos_w + kwNum;
                            if (pos_ori_w >= 0 && pos_ori_h >= 0 && pos_ori_w < w && pos_ori_h < h)
                            {
                                sum += (double)(in[n_num * c * h * w + c_num * (w * h) + pos_ori_h * w + pos_ori_w]);
                            }
                        }
                    }
                    out[n_num * c * out_h * out_w + c_num * out_h * out_w + i * out_w + j] = (float)(sum/(r*s));
                }
            }
        }
    }
}

void pooling_gpu(const int n,
                const int c,
                const int h,
                const int w,
                const int r,
                const int s,
                const int u,
                const int v,
                const int p,
                const int q,
                const int out_h, // 输出高
                const int out_w, // 输出宽
                float *in,
                float *out
){
    float *in_device, *out_device;

    hipMalloc((void **)&in_device, n * c * h * w * sizeof(float));
    hipMalloc((void **)&out_device, n * c * out_h * out_w * sizeof(float));

    // 将输入数据和卷积核拷贝到 GPU
    hipMemcpy(in_device, in, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out_device, out, n * c * out_h * out_w * sizeof(float), hipMemcpyHostToDevice);

    // 定义线程块的大小
    const int blockDim_x = 16;
    const int blockDim_y = 16;

    // 计算线程块和网格的数量
    const int gridDim_x = (out_h * out_w + blockDim_x - 1) / blockDim_x;
    const int gridDim_y = (c + blockDim_y - 1) / blockDim_y;

    // 定义线程块和网

    dim3 blockDim(blockDim_x, blockDim_y);
    dim3 gridDim(gridDim_x, gridDim_y, n);

    // 调用 kernel 函数
    naive_pooling_kernel<<<gridDim, blockDim>>>(n, c, h, w, r, s, out_h, out_w, u, v, p, q, in_device, out_device);
    // 同步
    hipDeviceSynchronize();

    // 将 GPU 计算的结果拷贝到 CPU
    hipMemcpy(out, out_device, n * c * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(in_device);
    hipFree(out_device);
}

vector<vector<vector<float>>> averagePooling3D_GPU(
    const vector<vector<vector<float>>> &inputMaps,
    const int poolsize,
    const int stride){

    const int n = 1;                           // batch size
    const int c = inputMaps.size();                           // 通道数
    const int h = inputMaps[0].size();                          // 数据高
    const int w = inputMaps[0][0].size();                          // 数据宽
    const int r = poolsize;                           // 卷积核高
    const int s = poolsize;                           // 卷积核宽
    const int u = stride;                           // 卷积在高方向上的步长
    const int v = stride;                           // 卷积在宽方向上的步长
    const int p = 0;                           // 卷积在高方向上的补边
    const int q = 0;                           // 卷积在宽方向上的补边
    const int out_h = (h + 2 * p - r) / u + 1; // 输出高
    const int out_w = (w + 2 * q - s) / v + 1; // 输出宽

    float *in, *out;
    in = (float *)malloc(n * c * h * w * sizeof(float));
    out = (float *)malloc(n * c * out_h * out_w * sizeof(float));

    int index = 0;
    for (int i = 0; i < c; ++i){
        for (int j=0; j<h;j++){
            for (int k=0; k<w;k++){
                in[index++] = inputMaps[i][j][k];
            }
        }
    }
    pooling_gpu(n,                           // batch size
                c,                          // 通道数
                h,                          // 数据高
                w,                          // 数据宽
                r,                           // 卷积核高
                s,                           // 卷积核宽
                u,                           // 卷积在高方向上的步长
                v,                           // 卷积在宽方向上的步长
                p,                           // 卷积在高方向上的补边
                q,                           // 卷积在宽方向上的补边
                out_h,                       // 输出高
                out_w,                       // 输出宽
                in, out);
    // Initialize output maps
    vector<vector<vector<float>>> outputMaps(c, vector<vector<float>>(out_h, vector<float>(out_w, 0.0)));
    index = 0;
    for (int i = 0; i < k; ++i){
        for (int j=0; j<out_h;j++){
            for (int k=0; k<out_w;k++){
                outputMaps[i][j][k] = out[index++];
            }
        }
    }
    free(in);
    free(out);
    return outputMaps;
}

int main()
{
    // 定义输入数据和卷积核的尺寸
    const int n = 1;                           // batch size
    const int c = 6;                           // 通道数
    const int h = 28;                          // 数据高
    const int w = 28;                          // 数据宽
    const int r = 2;                           // 卷积核高
    const int s = 2;                           // 卷积核宽
    const int u = 2;                           // 卷积在高方向上的步长
    const int v = 2;                           // 卷积在宽方向上的步长
    const int p = 0;                           // 卷积在高方向上的补边
    const int q = 0;                           // 卷积在宽方向上的补边
    const int out_h = (h + 2 * p - r) / u + 1; // 输出高
    const int out_w = (w + 2 * q - s) / v + 1; // 输出宽
    // 分配内存并随机生成输入数据和卷积核
    float *in, *out;
    in = (float *)malloc(n * c * h * w * sizeof(float));
    out = (float *)malloc(n * c * out_h * out_w * sizeof(float));
    // 随机生成输入数据和卷积核
    for (int i = 0; i < n * c * h * w; ++i)
    {
        in[i] = (float)rand() / RAND_MAX;
    }
    pooling_gpu(n,                           // batch size
                c,                          // 通道数
                h,                          // 数据高
                w,                          // 数据宽
                r,                           // 卷积核高
                s,                           // 卷积核宽
                u,                           // 卷积在高方向上的步长
                v,                           // 卷积在宽方向上的步长
                p,                           // 卷积在高方向上的补边
                q,                           // 卷积在宽方向上的补边
                out_h,                       // 输出高
                out_w,                       // 输出宽
                in, out);

    // CPU 端进行卷积计算
    float *out_cpu = (float *)malloc(n * c * out_h * out_w * sizeof(float));
    pooling_cpu(in, out_cpu, n, c, h, w, r, s, u, v, p, q, out_h, out_w);

    // 比较 GPU 和 CPU 计算结果是否一致
    bool pass = true;
    for (int i = 0; i < n * c * out_h * out_w; ++i)
    {
        if (abs(out[i] - out_cpu[i]) > 1e-5)
        {
            pass = false;
            std::cout << "Verification failed at " << i << "!" << std::endl;
            std::cout << "GPU: " << out_cpu[i] << " CPU: " << out[i] << std::endl;
            break;
        }
    }
    std::cout << "Verification Pass"<< std::endl;
    free(in);
    free(out);

    return 0;
}