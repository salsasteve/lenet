#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <math.h>

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void sgemm_naive_cpu(float *A, float *B, float *bias, float *C, int M, int N, int K)
{
    for (int x = 0; x < M; x++)
    {
        for (int y = 0; y < N; y++)
        {
            float sum = 0.0f;
            for (int i = 0; i < K; i++)
            {
                sum += A[x * K + i] * B[i * N + y];
            }
            C[x * N + y] = tanhf(sum + bias[x * N + y]);
        }
    }
}

template <const int BM, const int BN, const int BK, const int TM>
__global__ void sgemm_blocktiling_1d_kernel(float *A, float *B, float *C, int M, int N, int K)
{
    // the output block that we want to compute in this threadblock
    const int c_row = blockIdx.y;
    const int c_col = blockIdx.x;

    // allocate shared memory for the input and output submatrices
    __shared__ float A_shared[BM * BK];
    __shared__ float B_shared[BK * BN];

    // the inner row & col that we're accessing in this thread
    const int thread_row = threadIdx.x / BN;
    const int thread_col = threadIdx.x % BN;

    // advance pointers to the starting positions
    A += c_row * BM * K;
    B += c_col * BN;
    C += c_row * BM * N + c_col * BN;

    // use to avoid out-of-bounds accesses
    int global_m_pos = c_row * BM * K;
    int global_n_pos = c_col * BN;
    const int m_size = M * K;
    const int n_size = N * K;

    const int A_inner_row = threadIdx.x / BK; // warp-level GMEM coalescing
    const int A_inner_col = threadIdx.x % BK;
    const int B_inner_row = threadIdx.x / BN; // warp-level GMEM coalescing
    const int B_inner_col = threadIdx.x % BN;

    // allocate thread-local cache for results in registerfile
    float thread_results[TM] = {0.0};

    // outer loop over block tiles
    for (int bk_idx = 0; bk_idx < K; bk_idx += BK)
    {
        // load the next block of the input matrices into shared memory
        A_shared[A_inner_row * BK + A_inner_col] = (global_m_pos + A_inner_row * K + A_inner_col < m_size) ? A[A_inner_row * K + A_inner_col] : 0.0f;
        B_shared[B_inner_row * BN + B_inner_col] = (global_n_pos + B_inner_row * N + B_inner_col < n_size) ? B[B_inner_row * N + B_inner_col] : 0.0f;

        // wait for all threads to finish loading
        __syncthreads();

        // advance the pointers
        A += BK;
        B += BK * N;
        global_m_pos += BK;
        global_n_pos += BK * N;

        // compute the partial sum
        for (int dot_idx = 0; dot_idx < BK; dot_idx++)
        {
            // we make the dotproduct loop the outside loop, which facilitates
            // reuse of the Bs entry, which we can cache in a tmp var.
            float tmp_b = B_shared[dot_idx * BN + thread_col];
            for (int res_idx = 0; res_idx < TM; res_idx++)
            {
                thread_results[res_idx] += A_shared[(thread_row * TM + res_idx) * BK + dot_idx] * tmp_b;
            }
        }

        // wait for all threads to finish computing
        __syncthreads();
    }

    for (int res_idx = 0; res_idx < TM; res_idx++)
    {
        if (c_row * BM + thread_row * TM + res_idx < M && c_col * BN + thread_col < N)
        {
            C[(thread_row * TM + res_idx) * N + thread_col] = thread_results[res_idx];
        }
    }
}

__global__ void add_kernel(float *x, float *y, int n){
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if(tid < n) {
        x[tid] = tanhf(x[tid] + y[tid]);
    }
}


void run_sgemm_blocktiling_1d(float *A, float *B, float *bias, float *C, int m, int n, int k)
{
    const int BM = 64;
    const int BN = 64;
    const int BK = 8;
    const int TM = 8;
    dim3 grid_size(CEIL_DIV(n, BN), CEIL_DIV(m, BM));
    dim3 block_size((BM * BN) / TM);
    sgemm_blocktiling_1d_kernel<BM, BN, BK, TM>
        <<<grid_size, block_size>>>(A, B, C, m, n, k);
    const int block = 256;
    const int grid = (n + block) / block;
    add_kernel<<<grid, block>>>(C, bias, n);
}

void randomize_matrix(float *mat, int N)
{
    for (int i = 0; i < N; i++)
    {
        mat[i] = rand() % 100;
    }
}

std::vector<float> dense_GPU(
    std::vector<float> &input,
    std::vector<float> &bias,
    std::vector<std::vector<float>> &weights,
    int numOutputs)
{
    int m = 1;
    int n = numOutputs;
    int k = weights.size();

    // Allocate memory for matrices
    float *A, *B, *C, *bias;
    float *d_A, *d_B, *d_C, *d_bias;

    A = new float[m * k];
    B = new float[k * n];
    C = new float[m * n];
    bias = new float[m * n];
    int index = 0;
    for (int i = 0; i < k; ++i){
        A[index++] = input[i];
    }
    index = 0;
    for (int i = 0; i < k; ++i){
        for (int j=0; j<n;j++){
            B[index++] = weights[i][j];
        }
    }
    index = 0;
    for (int i = 0; i < n; ++i){
        bias[index++]=biases[i];
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));
    hipMalloc((void **)&d_bias, m * n * sizeof(float));

    // Copy matrices to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, m * n * sizeof(float), hipMemcpyHostToDevice);

    run_sgemm_blocktiling_1d(d_A, d_B, d_bias, d_C, m, n, k);

    // Copy result to host
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    std::vector<float> output(n);
    index = 0;
    for (int i = 0; i < n; ++i){
        outputMaps[i] = out[index++];
    }
    return output;
}

int main()
{
    int m = 1;
    int n = 120;
    int k = 400;

    // Allocate memory for matrices
    float *A, *B, *C, *C_ref, *bias;
    float *d_A, *d_B, *d_C, *d_bias;

    A = new float[m * k];
    B = new float[k * n];
    C = new float[m * n];
    bias = new float[m * n];
    // save reference result
    C_ref = new float[m * n];

    // Initialize matrices
    randomize_matrix(A, m * k);
    randomize_matrix(B, k * n);
    randomize_matrix(bias, m*n);

    // Allocate device memory
    hipMalloc((void **)&d_A, m * k * sizeof(float));
    hipMalloc((void **)&d_B, k * n * sizeof(float));
    hipMalloc((void **)&d_C, m * n * sizeof(float));
    hipMalloc((void **)&d_bias, m * n * sizeof(float));

    // Copy matrices to device
    hipMemcpy(d_A, A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, k * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_C, C, m * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, m * n * sizeof(float), hipMemcpyHostToDevice);

    run_sgemm_blocktiling_1d(d_A, d_B, d_bias, d_C, m, n, k);

    // Copy result to host
    hipMemcpy(C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Run reference sgemm
    sgemm_naive_cpu(A, B, bias, C_ref, m, n, k);

    // Verify result
    for (int i = 0; i < m * n; i++)
    {
        if (C[i] != C_ref[i])
        {
            printf("Error: mismatch at index %d, expected %f, got %f\n", i, C_ref[i], C[i]);
            return 1;
        }
    }
    int iter = 100;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    for (int i = 0; i < iter; i++){
        run_sgemm_blocktiling_1d(d_A, d_B, d_bias, d_C, m, n, k);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    std::cout << "GPU time: " << 1000 * elapsedTime / iter << "us" << std::endl;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("Success!\n");
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_bias);
    free(A);
    free(B);
    free(C);
    free(C_ref);
    free(bias);
    return 0;
}