#include "hip/hip_runtime.h"
#include <cassert>
#include <cstdlib>
#include <iostream>
#include "weights_and_bias.h"

/ 5 x 5 convolutional mask
#define MASK_DIM 5
#define DEPTH 1
#defince COUNT 6

// Amount the the matrix will hang over the matrix
#define MASK_OFFSET (MASK_DIM / 2)

// Allocate mask in constant memory
__constant__ int mask[6*1*5*5];

// 2D Convolution Kernel
// Takes:
//  matrix: Input matrix
//  result: Convolution result
//  N:      Dimensions of the matrices
__global__ void convolution_2d(int *matrix, int *result, int N, int depth, int count) {
  // Calculate the global thread positions
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  // Starting index for calculation
  int start_r = row - MASK_OFFSET;
  int start_c = col - MASK_OFFSET;

  // Temp value for accumulating the result
  int temp = 0;

  // Iterate over all the rows
  for (int i = 0; i < MASK_DIM; i++) {
    // Go over each column
    for (int j = 0; j < MASK_DIM; j++) {
      // Range check for rows
      if ((start_r + i) >= 0 && (start_r + i) < N) {
        // Range check for columns
        if ((start_c + j) >= 0 && (start_c + j) < N) {
          // Accumulate result
          temp += matrix[depth * N * N + (start_r + i) * N + (start_c + j)] *
                  mask[count * DEPTH * MASK_DIM * MASK_DIM + depth * MASK_DIM * MASK_DIM + i * MASK_DIM + j];
        }
      }
    }
  }

  // Write back the result
  result[count*N*N + row * N + col] = temp;
}

// Initializes an n x n matrix with random numbers
// Takes:
//  m : Pointer to the matrix
//  n : Dimension of the matrix (square)
void init_matrix(int *m, int n) {
  for (int i = 0; i < n; i++) {
    for (int j = 0; j < n; j++) {
      m[n * i + j] = rand() % 100;
    }
  }
}

// Verifies the 2D convolution result on the CPU
// Takes:
//  m:      Original matrix
//  mask:   Convolutional mask
//  result: Result from the GPU
//  N:      Dimensions of the matrix
void verify_result(int *m, int *mask, int *result, int N) {
  // Temp value for accumulating results
  int temp;

  // Intermediate value for more readable code
  int offset_r;
  int offset_c;

  // Go over each row
  for (int c = 0; c < COUNT; c++) {
    for (int d = 0; d < DEPTH; d++) {
      for (int i = 0; i < N; i++) {
        // Go over each column
        for (int j = 0; j < N; j++) {
          // Reset the temp variable
          temp = 0;

          // Go over each mask row
          for (int k = 0; k < MASK_DIM; k++) {
            // Update offset value for row
            offset_r = i - MASK_OFFSET + k;

            // Go over each mask column
            for (int l = 0; l < MASK_DIM; l++) {
              // Update offset value for column
              offset_c = j - MASK_OFFSET + l;
              // Range checks if we are hanging off the matrix
              if (offset_r >= 0 && offset_r < N) {
                if (offset_c >= 0 && offset_c < N) {
                  // Accumulate partial results
                  temp += m[d * N * N + offset_r * N + offset_c] * mask[c * DEPTH * MASK_DIM * MASK_DIM + DEPTH * MASK_DIM * MASK_DIM + k * MASK_DIM + l];
                }
              }
            }
          }
          // Fail if the results don't match
          assert(result[i * N + j] == temp);
        }
      }
    }
  }
}

int main() {
  // Dimensions of the matrix (2 ^ 10 x 2 ^ 10)
  int N = 1 << 10;

  // Size of the matrix (in bytes)
  size_t input_bytes_n = DEPTH * N * N * sizeof(int);
  size_t output_bytes_n = COUNT * N * N * sizeof(int);
  // Allocate the matrix and initialize it
  int *matrix = new int[DEPTH*N * N];
  int *result = new int[COUNT*N * N];
  init_matrix(matrix, N);

  // Size of the mask in bytes
  size_t bytes_m = DEPTH*COUNT*MASK_DIM * MASK_DIM * sizeof(int);

  // Allocate the mask and initialize it
  int *h_mask = new int[MASK_DIM * MASK_DIM];
                                                                                                                                                                                              123,0-1       62%
  init_matrix(h_mask, MASK_DIM);

  // Allocate device memory
  int *d_matrix;
  int *d_result;
  hipMalloc(&d_matrix, input_bytes_n);
  hipMalloc(&d_result, output_bytes_n);

  // Copy data to the device
  hipMemcpy(d_matrix, matrix, bytes_n, hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(mask), h_mask, bytes_m);

  // Calculate grid dimensions
  int THREADS = 16;
  int BLOCKS = (N + THREADS - 1) / THREADS;

  // Dimension launch arguments
  dim3 block_dim(THREADS, THREADS);
  dim3 grid_dim(BLOCKS, BLOCKS);
  for (int i = 0; i < DEPTH; i++) {
    for (int j = 0; j < COUNT; j++) {
      // Perform 2D Convolution
      convolution_2d<<<grid_dim, block_dim>>>(d_matrix, d_result, N, i, j);
    }
  }

  // Copy the result back to the CPU
  hipMemcpy(result, d_result, bytes_n, hipMemcpyDeviceToHost);

  // Functional test
  verify_result(matrix, h_mask, result, N);

  std::cout << "COMPLETED SUCCESSFULLY!";

  // Free the memory we allocated
  delete[] matrix;
  delete[] result;
  delete[] h_mask;

  hipFree(d_matrix);
  hipFree(d_result);

  return 0;
}
